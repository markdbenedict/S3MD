#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <signal.h>
#include <sys/time.h>

typedef float real;

#define CHAR_MINUS  '-'
#define CHAR_ZERO   '0'

typedef struct {
  real u[9];
} RMat;

#define MAT(a, n, i, j)  (a)[(i) + n * (j)]

#define AllocMem(a, n, t)  a = (t *) malloc ((n) * sizeof (t))

#define AllocMem2(a, n1, n2, t)                             \
   AllocMem (a, n1, t *);                                   \
   AllocMem (a[0], (n1) * (n2), t);                         \
   for (k = 1; k < n1; k ++) a[k] = a[k - 1] + n2;

#define MAX_MPEX_ORD  2
#define I(i, j)  ((i) * ((i) + 1) / 2 + (j))
#define c(i, j)  c[I(i, j)]
#define s(i, j)  s[I(i, j)]

typedef struct {
  real c[I(MAX_MPEX_ORD, MAX_MPEX_ORD) + 1], s[I(MAX_MPEX_ORD, MAX_MPEX_ORD) + 1];
} MpTerms;
typedef struct {
  MpTerms le, me;
  int occ;
} MpCell;

#include "in_vdefs.h"
#include "in_namelist.h"
#include "in_proto.h"

#define DO_MOL  for (n = 0; n < nMol; n ++)
#define DO_CELL(j, m)  for (j = cellList[m]; j >= 0; j = cellList[j])

#define VWrap(v, t)                                         \
   if (v.t >= 0.5 * (*inRegion).t)      v.t -= (*inRegion).t;         \
   else if (v.t < -0.5 * (*inRegion).t) v.t += (*inRegion).t

#define VShift(v, t)                                        \
   if (v.t >= 0.5 * (*inRegion).t)      shift.t -= (*inRegion).t;     \
   else if (v.t < -0.5 * (*inRegion).t) shift.t += (*inRegion).t

#define VShiftWrap(v, t)                                    \
   if (v.t >= 0.5 * (*inRegion).t) {                             \
     shift.t -= (*inRegion).t;                                   \
     v.t -= inRegion.t;                                       \
   } else if (v.t < -0.5 * (*inRegion.t)) {                      \
     shift.t += (*inRegion.t);                                   \
     v.t += (*inRegion).t;                                       \
   }

#define VCellWrap(t)                                        \
   if (m2v.t >= cells.t) {                                  \
     m2v.t = 0;                                             \
     shift.t = inRegion.t;                                    \
   } else if (m2v.t < 0) {                                  \
     m2v.t = cells.t - 1;                                   \
     shift.t = - inRegion.t;                                  \
   }

#define VWrapAll(v)                                         \
   {VWrap (v, x);                                           \
   VWrap (v, y);                                            \
   VWrap (v, z);}
#define VShiftAll(v)                                        \
   {VShift (v, x);                                          \
   VShift (v, y);                                           \
   VShift (v, z);}
#define VCellWrapAll()                                      \
   {VCellWrap (x);                                          \
   VCellWrap (y);                                           \
   VCellWrap (z);}

#define OFFSET_VALS                                           \
   { {0,0,0}, {1,0,0}, {1,1,0}, {0,1,0}, {-1,1,0},            \
     {0,0,1}, {1,0,1}, {1,1,1}, {0,1,1}, {-1,1,1}, {-1,0,1},  \
     {-1,-1,1}, {0,-1,1}, {1,-1,1}                            \
   }

 
// Includes
#include <stdio.h>
#include <cutil_inline.h>

// Variables
typedef struct {
  VecR r, rv, ra, ra1, ra2, ro, rvo;
} Mol;



Mol     * d_mol;
VecR    *d_region;
int     *d_nebrTabPtr, *d_nebrTab;
real    *d_atomPotential,*d_g;
int     *d_indexSum;
real    *d_accum;
real    *h_atomPotential;
int     h_nMol;
VecI    h_UCell;
bool noprompt = false;

// Functions
void Cleanup(void);

void AllocGPUMemory(int nebrTabMax, int nebrTabLen,int nMol,VecR region,VecI inUnitCell)
{
    h_nMol = nMol;
    h_UCell=inUnitCell;
    hipDeviceProp_t prop;
    int Dev;
    hipGetDevice(&Dev);
    hipGetDeviceProperties(&prop,Dev);
    
    printf("ID of current CUDA Device = %d\n",Dev);
    printf("the name of device is %s\n",prop.name);
    printf("compute capability %d.%d\n",prop.major,prop.minor);
    printf("inside .cu nMol=%d\n\n",nMol);
    printf("h_nMol=%d\n\n",h_nMol);
    
    h_atomPotential=(real*)malloc(sizeof(real)*h_nMol);
    cutilSafeCall(hipMalloc((void**)&d_nebrTabPtr, (h_nMol+1)*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&d_nebrTab, nebrTabMax*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&d_mol, h_nMol*sizeof(Mol)));
    cutilSafeCall(hipMalloc((void**)&d_g, 15*h_nMol*sizeof(real)));
    cutilSafeCall(hipMalloc((void**)&d_atomPotential, h_nMol*sizeof(real)));
    cutilSafeCall(hipMalloc((void**)&d_indexSum,sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&d_region,sizeof(VecR)));
    cutilSafeCall(hipMemcpy(d_region, &region, sizeof(VecR), hipMemcpyHostToDevice));
    cutilSafeCall(hipMalloc((void**)&d_accum, h_nMol*sizeof(real)));    
}

void UpdateGPUNeighbors(int* nebrTabPtr,int* nebrTab,int nebrTabMax)
{
    hipMemcpy(d_nebrTabPtr, nebrTabPtr, (h_nMol+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nebrTab, nebrTab, nebrTabMax*sizeof(int), hipMemcpyHostToDevice);
}

__global__ void ZeroAccumulators(Mol* inMol,real* inAtomPotential,int inNumMol,real* accum,int* inCount)
{
  int n=blockIdx.x;//8*gridDim.x *gridDim.x * blockIdx.x + 8*gridDim.y*blockIdx.y + threadIdx.x;
  if(n < inNumMol)
  {
    atomicAdd(inCount,n);
    inMol[n].ra.x=0.;
    inMol[n].ra.y=0.;
    inMol[n].ra.z=0.;
    inAtomPotential[n]=0.;
    accum[n]=0.;
    
  }
}

__global__ void ComputeForcesGPU(Mol *inMol,real* inAtomPotential,int* inNebrTabPtr,int*inNebrTab,VecR* inRegion,real inRCut,int inNMol,int* inCounter,real* accum)
{
    VecR dr, dr12, dr13, w2, w3;
    real aCon = 7.0496, bCon = 0.60222, cr, er, fcVal,
       gCon = 1.2, lCon = 21., p12, p13, ri, ri3, rm,
       rm12, rm13, rm23,rr, rr12, rr13, rrCut;
    int j2, j3, m2, m3;
    int CURR=blockIdx.x;
    real fc=0,theta=0;
    real eta[5] = {0.01,0.1,0.5,1.0,10.0};
    real Rs[6] = {1.0,2.0,3.0,4.0,5.0,6.0};
    real g[15] ={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    int zeta[3] = {1,2,3};
    int lambda[2] = {-1,1};
    if(CURR<inNMol)
    {
        atomicAdd(inCounter,CURR);
        accum[CURR]=inRCut;
        rrCut = Sqr (inRCut) - 0.001;
        for (m2 = inNebrTabPtr[CURR]; m2 < inNebrTabPtr[CURR + 1]; m2 ++)
        {
            j2 = inNebrTab[m2];
            if (CURR<j2)
            {
                //atomicMax(inCounter,j2);   
                VSub (dr, inMol[CURR].r, inMol[j2].r);
                VWrapAll (dr);
                rr = VLenSq (dr);
                if (rr < rrCut) {
                     //calculate if i,j conribution to G1(i)and add it to sum for i
                     rm = sqrt (rr);
                     er = exp (1. / (rm - inRCut));
                     ri = 1. / rm;
                     ri3 = Cube (ri);
                     fcVal = aCon * (4. * bCon * Sqr (ri3) +
                                         (bCon * ri3 * ri - 1.) * ri / Sqr (rm - inRCut)) * er;
                     VVSAdd (inMol[CURR].ra, fcVal, dr);
                     VVSAdd (inMol[j2].ra, - fcVal, dr);
                     //uSum += aCon * (bCon * ri3 * ri - 1.) * er;
                     inAtomPotential[CURR]+=aCon * (bCon * ri3 * ri - 1.) * er;
                     fc=1.0+0.5*cos(3.1419*rm/inRCut);
                     g[0]+=exp(-eta[0]*(rm-Rs[0])*(rm-Rs[0]))*fc;
                     g[1]+=exp(-eta[0]*(rm-Rs[5])*(rm-Rs[5]))*fc;
                     
                     g[2]+=exp(-eta[1]*(rm-Rs[0])*(rm-Rs[0]))*fc;
                     g[3]+=exp(-eta[4]*(rm-Rs[1])*(rm-Rs[1]))*fc;
                     g[4]+=exp(-eta[1]*(rm-Rs[2])*(rm-Rs[2]))*fc;
                     g[5]+=exp(-eta[1]*(rm-Rs[4])*(rm-Rs[4]))*fc;
                     g[6]+=exp(-eta[1]*(rm-Rs[5])*(rm-Rs[5]))*fc;
                     
                     g[7]+=exp(-eta[2]*(rm-Rs[0])*(rm-Rs[0]))*fc;
                     g[8]+=exp(-eta[2]*(rm-Rs[3])*(rm-Rs[3]))*fc;
                     
                }
            }
        }
	
        //3 body terms
        /*
        for (m2 = inNebrTabPtr[CURR]; m2 < inNebrTabPtr[CURR + 1] - 1; m2 ++)
        {
           j2 = inNebrTab[m2];
           VSub (dr12, inMol[CURR].r, inMol[j2].r);
           VWrapAll (dr12);
           rr12 = VLenSq (dr12);
           if (rr12 < rrCut)
           {
                rm12 = sqrt (rr12);
                VScale (dr12, 1. / rm12);
                for (m3 = m2 + 1; m3 < inNebrTabPtr[CURR + 1]; m3 ++)
                {
                     j3 = inNebrTab[m3];
                     VSub (dr13, inMol[CURR].r, inMol[j3].r);
                     VWrapAll (dr13);
                     rr13 = VLenSq (dr13);
                     if (rr13 < rrCut)
                     {
                          rm13 = sqrt (rr13);
                          VScale (dr13, 1. / rm13);
                          cr = VDot (dr12, dr13);
                          er = lCon * (cr + 1./3.) * exp (gCon / (rm12 - inRCut) + gCon /
                                                                  (rm13 - inRCut));
                          p12 = gCon * (cr + 1./3.) / Sqr (rm12 - inRCut);
                          p13 = gCon * (cr + 1./3.) / Sqr (rm13 - inRCut);
                          VSSAdd (w2, p12 + 2. * cr / rm12, dr12, - 2. / rm12, dr13);
                          VSSAdd (w3, p13 + 2. * cr / rm13, dr13, - 2. / rm13, dr12);
                          VScale (w2, - er);
                          VScale (w3, - er);
                          VVSub (inMol[CURR].ra, w2);
                          VVSub (inMol[CURR].ra, w3);
                          VVAdd (inMol[j2].ra, w2);
                          VVAdd (inMol[j3].ra, w3);
                          //uSum += (cr + 1./3.) * er;
                          inAtomPotential[CURR]+=(cr + 1./3.) * er;
                          theta=acos(cr/rm13/rm12);
                          VecR dr23;
                          VSub (dr23, inMol[j2].r, inMol[j3].r); 
                          rm23= sqrt(VLenSq (dr23));
                          fc=(1.0+0.5*cos(3.1419*rm12/inRCut))*(1.0+0.5*cos(3.1419*rm13/inRCut))*(1.0+0.5*cos(3.1419*rm23/inRCut));
                          g[9]+=pow((real)2,1-zeta[0])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[0]) * exp(-eta[0]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          g[10]+=pow((real)2,1-zeta[0])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[0]) * exp(-eta[1]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          g[11]+=pow((real)2,1-zeta[0])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[0]) * exp(-eta[2]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          g[12]+=pow((real)2,1-zeta[0])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[0]) * exp(-eta[3]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          g[13]+=pow((real)2,1-zeta[1])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[1]) * exp(-eta[2]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          g[14]+=pow((real)2,1-zeta[2])*pow((real)(1.0+lambda[0]*cos(theta)),zeta[2]) * exp(-eta[2]*(rm12*rm12+rm13*rm13+rm23*rm23)) * fc;
                          
                     }
                }
           }
        }*/
        
    }
    
}
 
/*
__device__ void zeroArray(double* theArray)
{
  

}*/




// Host code
void doForceIterartion(double inRCut,Mol* inMol,double &outUSum)
{
    //real g1Sum=0;
    //int n;
    int counter=0;
    real localUSum=outUSum;
    dim3 theSize(8000,1,1);
    //printf("h_nMol=%d\n",h_nMol);
    /*g1Sum=0;
    for(n=0;n<h_nMol;n++)
    {
        g1Sum+=mol[n].ra.x+mol[n].ra.y+mol[n].ra.z;
    }*/
    //printf("mol[n].ra sum just before zero=%6.4f\n",g1Sum);
    
    //clear out accumulators
    real sum=0;
    int n;
    real* theAccum=(real*)malloc(h_nMol*sizeof(real));
    cutilSafeCall(hipMemcpy(d_indexSum,&counter, sizeof(int), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_mol,inMol,h_nMol*sizeof(Mol), hipMemcpyHostToDevice));
    int blocks=8000;
    int threads=1;
    ZeroAccumulators<<<blocks,threads>>>(d_mol,d_atomPotential,8000,d_accum,d_indexSum);
    //cutilSafeCall(hipMemcpy(theAccum, d_accum, h_nMol*sizeof(real), hipMemcpyDeviceToHost));
    //cutilSafeCall(hipMemcpy(h_atomPotential, d_atomPotential, h_nMol*sizeof(real), hipMemcpyDeviceToHost));
    //hipMemcpy(&counter, d_indexSum, sizeof(int), hipMemcpyDeviceToHost);
    //printf("sum of iterators=%d\n",counter);
    //for(n=0;n<h_nMol;n++) sum+=theAccum[n];
    //printf("Accum after zero=%f with h_nMol=%d\n",sum/h_nMol,h_nMol);
    //localUSum=0;
    //for(n=0;n<h_nMol;n++) localUSum+=h_atomPotential[n];
    //printf("localUSum after zero=%f\n",localUSum/h_nMol);
    //outUSum=localUSum/h_nMol;
    //printf("uSum after zero=%f\n",outUSum);
    //hipDeviceSynchronize();
    cutilSafeCall(hipMemcpy(d_indexSum,&counter, sizeof(int), hipMemcpyHostToDevice));
    ComputeForcesGPU<<<blocks,threads>>>(d_mol,d_atomPotential, d_nebrTabPtr,d_nebrTab,d_region,inRCut,h_nMol,d_indexSum,d_accum);
    cutilSafeCall(hipMemcpy(inMol, d_mol, h_nMol*sizeof(Mol), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_atomPotential, d_atomPotential, h_nMol*sizeof(real), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(theAccum, d_accum, h_nMol*sizeof(real), hipMemcpyDeviceToHost));
    hipMemcpy(&counter, d_indexSum, sizeof(int), hipMemcpyDeviceToHost);
    printf("sum of iterations=%d\n",counter);
    for(n=0;n<h_nMol;n++) sum+=theAccum[n];
    printf("Accum after zero=%f with h_nMol=%d\n",sum/h_nMol,h_nMol);
    localUSum=0;
    for(n=0;n<h_nMol;n++) localUSum+=h_atomPotential[n];
    printf("localUSum after zero=%f\n",localUSum/h_nMol);
    outUSum=localUSum/h_nMol;
    printf("uSum after zero=%f\n",outUSum);
  
    free(theAccum);

}

void Cleanup(void)
{
    // Free device memory
    //if (d_A)
    //    hipFree(d_A);
        
    //hipDeviceReset();
    /*
    if (!noprompt) {
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();
    }*/

    //exit(0);
}



